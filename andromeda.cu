#include "hip/hip_runtime.h"
/*
 *  This program is a CUDA C program simulating the N-body system
 *    of two galaxies as PHY 241 FINAL PROJECTS
 *
 */

/*
 *  TODO:(*for final project)
 *    1. andromeda
 *    2. report
 *    3. presentation
 *	  *4. N-body galaxy code-generat 10^11 particles
 *	  *5. MatLab write a function to track the distance between Milkway and Andromeda
 *	  *6. change accel function to the N-body one.
 *	  *7. print mass[i]. because the halo is dark matter. Or better way distinguish dark matter and rings?
 */


#include <hip/hip_runtime.h>
#include <math.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "dataDeclaration.h"
#include "functionDeclaration.h"
#include "otherfunctions.c"


/**     Main function     **/
int main(int argc, char *argv[])
{
  /*
   *  Handling commandline inputs and setting initial value of the arguments
   *    1. number of steps (mstep)
   *    2. warp (nout)
   *    3. offset (start printing position)
   *    4. timestamp (dt)
   *
   */
  int mstep, nout, offset, tnow = 0, n;
  double dt, *x, *y, *z, *vx, *vy, *vz, *mass;
  mstep = argc > 1 ? atoi(argv[1]) : 100;
  nout = argc > 2 ? atoi(argv[2]) : 1;
  offset = argc > 3 ? atoi(argv[3]) : 0;
  dt = argc > 4 ? atof(argv[4]) : 2 * PI * RMIN * RMIN /sqrt(G * MASS_1) / 200.0;
//   dt = argc > 4 ? atof(argv[4]) : 0.1;
  initialCondition_host_file("milky_way.dat", "andromeda.dat", &x, &y, &z, &vx, &vy, &vz, &mass, &n);
  int grids = ceil((double)n / BLOCKSIZE), threads = BLOCKSIZE;
  /*
   *  Use hipDeviceSetLimit() to change the buffer size of printf
   *   used in kernel functions to solve the problem encountered before:
   *    cannot print more than 4096 lines of data using printf
   *
   */
  hipDeviceSetLimit(hipLimitPrintfFifoSize, n * BUFFERSIZE);

  /*  Start looping steps from first step to mstep  */
  for(int i = 0; i < offset; i++, tnow++){
    hipDeviceSynchronize();
    accel<<< grids, threads >>>(n, x, y, z, vx, vy, vz, mass, dt);
    hipDeviceSynchronize();
    leapstep<<< grids, threads >>>(n, x, y, z, vx, vy, vz, dt);
    hipDeviceSynchronize();
    accel<<< grids, threads >>>(n, x, y, z, vx, vy, vz, mass, dt);
    hipDeviceSynchronize();
  }
  for(int i = offset; i < mstep; i++, tnow++){
    if(i % nout == 0)
      printstate<<< grids, threads >>>(n, x, y, z, vx, vy, vz, tnow);
    hipDeviceSynchronize();

    /*
     *  Use hipDeviceSynchronize() to wait till all blocks of threads
     *   finish running the kernel functions
     *  Since between each accel() is called, the position of each particle
     *   is updated, which affect the second accel() calls, we need sychronize
     *   in the middle
     *
     */
    accel<<< grids, threads >>>(n, x, y, z, vx, vy, vz, mass, dt);
    hipDeviceSynchronize();
    leapstep<<< grids, threads >>>(n, x, y, z, vx, vy, vz, dt);
    hipDeviceSynchronize();
    accel<<< grids, threads >>>(n, x, y, z, vx, vy, vz, mass, dt);
    hipDeviceSynchronize();
  }
  if(mstep % nout == 0)
    printstate<<< grids, threads >>>(n, x, y, z, vx, vy, vz, tnow);
  hipDeviceSynchronize();

  // After finishing, free the allocated memory
  hipFree(x);
  hipFree(y);
  hipFree(z);
  hipFree(vx);
  hipFree(vy);
  hipFree(vz);
  hipFree(mass);

  // Exit the current thread
  hipDeviceReset();
  return 0;
}

/*
 *  Functions Implmenetation Section
 *
 */
__global__ void initialConditions(int n, double* x, double* y, double* z, double* vx, double* vy, double* vz, double* mass){
  /*  TODO    */
}

void rotate(double* x, double* y, double *z, double n1, double n2, double n3, double theta){

   double tmpx, tmpy, tmpz;
   double a, c, s, sigma;

   sigma = -theta;
   c = cos(sigma);
   s = sin(sigma);
   a = 1 - cos(sigma);


  tmpx = ( a * n1 * n1 + c ) * (*x) + ( a * n1 * n2 - s * n3) * (*y) + ( a * n1 * n3 + s * n2 ) * (*z);
  tmpy = ( a * n1 * n2 + s * n3) * (*x) + ( a * n2 * n2 + c) * (*y) + ( a * n2 * n3 - s * n1 ) * (*z);
  tmpz = ( a * n1 * n3 - s * n2) * (*x) + ( a * n2 * n3 + s * n1) * (*y) + ( a * n3 * n3 + c) * (*z);

  (*x) = tmpx;
  (*y) = tmpy;
  (*z) = tmpz;

}

__global__ void leapstep(int n, double *x, double *y, double *z, double *vx, double *vy, double *vz, double dt){
  const unsigned int serial = blockIdx.x * BLOCKSIZE + threadIdx.x;
  if(serial < n){
    x[serial] += dt * vx[serial];
    y[serial] += dt * vy[serial];
    z[serial] += dt * vz[serial];
  }
}


__global__ void accel(int n, double *x, double *y, double *z, double *vx, double *vy, double *vz, double* mass, double dt){
  const unsigned int serial = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int tdx = threadIdx.x;
  __shared__ double lx[BLOCKSIZE];
  __shared__ double ly[BLOCKSIZE];
  __shared__ double lz[BLOCKSIZE];
  __shared__ double lm[BLOCKSIZE];

  double ax = 0.0, ay = 0.0, az = 0.0, norm, thisX = x[serial], thisY = y[serial], thisZ = z[serial];
  for(int i = 0; i < gridDim.x; i++){
    // Copy data from main memory
    lx[tdx] = x[i * blockDim.x + tdx];
    lz[tdx] = y[i * blockDim.x + tdx];
    ly[tdx] = z[i * blockDim.x + tdx];
    __syncthreads();
    // Accumulates the acceleration
    for(int j = 0; j < blockDim.x; j++){
      norm = pow(SOFTPARAMETER + pow(thisX - lx[j], 2) + pow(thisY - ly[j], 2) + pow(thisZ - lz[j], 2), 1.5);
      if(i * BLOCKSIZE + j != serial){
        ax += - G * lm[i * blockDim.x + j] * (thisX - lx[j]) / norm;
        ay += - G * lm[i * blockDim.x + j] * (thisY - ly[j]) / norm;
        az += - G * lm[i * blockDim.x + j] * (thisZ - lz[j]) / norm;
      }
    }
  }
  if(serial < n){
    vx[serial] += 0.5 * dt * ax;
    vy[serial] += 0.5 * dt * ay;
    vz[serial] += 0.5 * dt * az;
  }
}

__global__ void printstate(int n, double *x, double *y, double *z, double *vx, double *vy, double *vz, int tnow){
  const unsigned int serial = blockIdx.x * blockDim.x + threadIdx.x;
  if(serial < n){
    printf("%d, %12.6f, %12.6f, %12.6f, %12.6f, %12.6f, %12.6f, %d\n", serial, x[serial], y[serial], z[serial], vx[serial], vy[serial], vz[serial], tnow);
  }
}

void initialCondition_host_file(char *input1, char *input2, double **x, double **y, double **z, double **vx, double **vy, double **vz, double **mass, int *size){
  FILE *fp1 = fopen(input1, "r");
  FILE *fp2 = fopen(input2, "r");
  if(fp1 == NULL || fp2 == NULL){
    printf("Error: fail to open a file.\n");
    exit(-1);
  }
  int s1, s2;
  double unknown;
  read_size_from_file(input1, &s1);
  (*size) = s1;
  read_size_from_file(input2, &s2);
  (*size) += s2;
  s1 = (*size) - s2;

  // Initial local data array
  double *lx, *ly, *lz, *lvx, *lvy, *lvz, *lm;
  lx = (double*)malloc((*size) * 7 * sizeof(double));
  ly = lx + (*size);
  lz = ly + (*size);
  lvx = lz + (*size);
  lvy = lvx + (*size);
  lvz = lvy + (*size);
  lm = lvz + (*size);

  // Read data from file1
  FILE *fp = fopen(input1, "r");
  if(fp == NULL){
    printf("Error: fail to open file 1\n");
    exit(-1);
  }
  int junk1;
  double junk2;
  int count = 0;
  fscanf(fp, "%lu %lf", &junk1, &junk2);    // skip first line
  while(!feof(fp)){
    fscanf(fp, "%lf %lf %lf %lf %lf %lf %lf", lm + count, lx + count, ly + count, lz + count, lvx + count, lvy + count, lvz + count);
    *(lx + count) += MilkwayXOffset;
    *(ly + count) += MilkwayYOffset;
    *(lz + count) += MilkwayZOffset;
    count++;
  }
  fclose(fp);

  // Read data from file2
  fp = fopen(input2, "r");
  if(fp == NULL){
    printf("Error: fail to open file 2\n");
    exit(-1);
  }
  fscanf(fp, "%lu %lf", &junk1, &junk2);    // skip first line
  while(!feof(fp)){
    fscanf(fp, "%lf %lf %lf %lf %lf %lf %lf", lm + count, lx + count, ly + count, lz + count, lvx + count, lvy + count, lvz + count);
    *(lx + count) += AndromedaXOffset;
    *(ly + count) += AndromedaYOffset;
    *(lz + count) += AndromedaZOffset;
    count++;
  }
  fclose(fp);
  // Allocate device memory
  int numOfBlocks = ceil((double)(*size) / BLOCKSIZE);
  hipMalloc((void**)x, numOfBlocks * BLOCKSIZE * 7 * sizeof(double));
  *y = *x + numOfBlocks * BLOCKSIZE;
  *z = *y + numOfBlocks * BLOCKSIZE;
  *vx = *z + numOfBlocks * BLOCKSIZE;
  *vy = *vx + numOfBlocks * BLOCKSIZE;
  *vz = *vy + numOfBlocks * BLOCKSIZE;
  *mass = *vz + numOfBlocks * BLOCKSIZE;
  hipMemcpy((void**)x, lx, numOfBlocks * BLOCKSIZE * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy((void**)y, ly, numOfBlocks * BLOCKSIZE * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy((void**)z, lz, numOfBlocks * BLOCKSIZE * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy((void**)vx, lvx, numOfBlocks * BLOCKSIZE * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy((void**)vy, lvy, numOfBlocks * BLOCKSIZE * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy((void**)vz, lvz, numOfBlocks * BLOCKSIZE * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy((void**)mass, lm, numOfBlocks * BLOCKSIZE * sizeof(double), hipMemcpyHostToDevice);
  free(lx);
  fclose(fp1);
  fclose(fp2);
}

void read_size_from_file(char *input, int *size){
  FILE *fp = fopen(input, "r");
  double unknown;
  fscanf(fp, "%lu", size);
  (*size)++;
  fclose(fp);
}
