#include "hip/hip_runtime.h"
/*
 *  This program is a CUDA C program simulating the N-body system
 *    of two galaxies as PHY 241 FINAL PROJECTS
 *
 */

/*
 *  TODO:
 *    1. andromeda
 *    2. For accel of center of A, only consider accel from center of B. The same for B.
 *    3. When the distance between A and B, the soft parameter changed to 0.2Rmin
 *    4. report
 *    5. presentation
 *
 */


#include <hip/hip_runtime.h>
#include <math.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

/*
**  Modify the constant parameters if neccessary
**    Constant Section
*/
#define PI 3.14159265
#define BUFFERSIZE 256
#ifndef BLOCKSIZE
  #define BLOCKSIZE 256
#endif
//#define SOFTPARAMETER 0.2 * RMIN
// #define AU 149597870700.0
// #define R (77871.0 * 1000.0 / AU)
// #define G (4.0 * pow(PI, 2))
#define G 0.287915013
#define MASS_1 1000              // Center mass of 1st galaxy
#define MASS_2 1000                // Center mass of 2nd galaxy
#define NUM_OF_RING_1 12         // Number of rings in 1st galaxy
#define NUM_OF_RING_2 12          // Number of rings in 2nd galaxy
// #define RING_BASE_1 (R * 0.2)       // Radius of first ring in 1st galaxy
// #define RING_BASE_2 (R * 0.2)       // Radius of first ring in 2nd galaxy
#define NUM_P_BASE 12             // Number of particles in the first ring
#define INC_NUM_P 3               // increment of number of particles each step
// #define INC_R_RING (0.5 * R)      // increment of radius of rings each step
#define PMASS 1             // mass of each particle
#define V_PARAMTER 1            // Parameter adding to initial velocity to make it elliptic
#define RMIN 1
#define ECCEN 0.5
#define RMAX ((1.0 + ECCEN) * RMIN / (1.0 - ECCEN))
#define RING_BASE_1 (RMIN * 0.2)       // Radius of first ring in 1st galaxy
#define RING_BASE_2 (RMIN * 0.2)       // Radius of first ring in 2nd galaxy
#define INC_R_RING (RMIN * 0.05)      // increment of radius of rings each step
#define SOFTPARAMETER 0.000001
/*
 *  Major Function Declarations Section
 *
 */

__global__ void leapstep(int, double*, double*, double*, double*, double*, double*, double);
/*
 *  leapstep - kernel function: update positions using leapfrog algorithm
 *    parameters:
 *      [#particles] [x position] [y position] [x position]
 *                   [x velocity] [y velocity] [z velocity] [dt]
 *
 */
__global__ void accel(int, double*, double*, double*, double*, double*, double*, double*, double);
/*
 *  accel - kernel function: update velocity using leapfrog algorithm
 *    parameters:
 *      [#particles] [x position] [y position] [x position]
 *                   [x velocity] [y velocity] [z velocity]
 *                   [mass] [dt]
 *
 */
__global__ void accel_3_body(int, double*, double*, double*, double*, double*, double*, double*, double);
/*
 *  accel_3_body - kernel function : update velocity using leapfrog algorithm in 3-body
 *    parameters:
 *      [#particles] [x position] [y position] [x position]
 *                   [x velocity] [y velocity] [z velocity]
 *                   [mass] [dt]
 *
 */
__global__ void printstate(int, double*, double*, double*, double*, double*, double*);
/*
 *  printstate - kernel function: print position and velocity
 *    parameters:
 *      [#particles] [x position] [y position] [x position]
 *                   [x velocity] [y velocity] [z velocity]
 *
 */
void printstate_host(int, double*, double*, double*, double*, double*, double*);
/*
 *  printstate_host - host function: print position and velocity
 *    parameters:
 *      [#particles] [x position] [y position] [x position]
 *                   [x velocity] [y velocity] [z velocity]
 *
 */
__global__ void initialConditions(int, double*, double*, double*, double*, double*, double*, double*);
/*
 *  initialConditions - kernel function: setup initial conditions
 *    parameters:
 *      [#particles] [x position] [y position] [x position]
 *                   [x velocity] [y velocity] [z velocity] [mass]
 *
 */
void initialCondition_host(int, double*, double*, double*, double*, double*, double*, double*);
/*
 *  initialCondition_host - host function: setup initial conditions
 *    parameters:
 *      [#particles] [x position] [y position] [x position]
 *                   [x velocity] [y velocity] [z velocity] [mass]
 *
 */

/*
 *  Helper Function Declarations
 *
 */
void rotate(double*, double*, double*, double, double, double, double);

/**     Main function     **/
int main(int argc, char *argv[])
{
  /*
   *  Handling commandline inputs and setting initial value of the arguments
   *    1. number of steps (mstep)
   *    2. warp (nout)
   *    3. timestamp (dt)
   *
   */
  int mstep, nout;
  double dt, *x, *y, *z, *vx, *vy, *vz, *mass;
  mstep = argc > 1 ? atoi(argv[1]) : 100;
  nout = argc > 2 ? atoi(argv[2]) : 20;
  dt = argc > 3 ? atof(argv[3]) : 2 * PI * RMIN * RMIN /sqrt(G * MASS_1) / 500.0;
  int n = (NUM_P_BASE + NUM_P_BASE + (NUM_OF_RING_1 - 1) * INC_NUM_P) * NUM_OF_RING_1 / 2 + (NUM_P_BASE + NUM_P_BASE + (NUM_OF_RING_2 - 1) * INC_NUM_P) * NUM_OF_RING_2 / 2 + 2;
  /*
   *  setup execution configuration
   */
  int numOfBlocks = n / BLOCKSIZE + (n % BLOCKSIZE != 0);
  int threads = BLOCKSIZE, grids = numOfBlocks;

  /*
  ** Allocate device memory for kernel functions
  **  May not need to allocate memory for host function
  **  because we print using kernel function
  ** Use numOfBlocks instead of n to simplify the kernel function
  */
  const unsigned int extra = numOfBlocks * BLOCKSIZE - n;
  hipMalloc((void**) &x, (size_t)(numOfBlocks * BLOCKSIZE * sizeof(double)));
  hipMalloc((void**) &y, (size_t)(numOfBlocks * BLOCKSIZE * sizeof(double)));
  hipMalloc((void**) &z, (size_t)(numOfBlocks * BLOCKSIZE * sizeof(double)));
  hipMalloc((void**) &vx, (size_t)(numOfBlocks * BLOCKSIZE * sizeof(double)));
  hipMalloc((void**) &vy, (size_t)(numOfBlocks * BLOCKSIZE * sizeof(double)));
  hipMalloc((void**) &vz, (size_t)(numOfBlocks * BLOCKSIZE * sizeof(double)));
  hipMalloc((void**) &mass, (size_t)(numOfBlocks * BLOCKSIZE * sizeof(double)));
  hipMemset((void**) x + n, 0, (size_t)(extra * sizeof(double)));
  hipMemset((void**) y + n, 0, (size_t)(extra * sizeof(double)));
  hipMemset((void**) z + n, 0, (size_t)(extra * sizeof(double)));
  hipMemset((void**) vx + n, 0, (size_t)(extra * sizeof(double)));
  hipMemset((void**) vy + n, 0, (size_t)(extra * sizeof(double)));
  hipMemset((void**) vz + n, 0, (size_t)(extra * sizeof(double)));
  hipMemset((void**) mass + n, 0, (size_t)(extra * sizeof(double)));

  /*
   *  If MCORE is defined, use kernel function to setup
   *    initial conditions
   *  Otherwise, use host function to setup initial conditions
   *
   */
#ifdef MCORE
  initialConditions<<< grids, threads >>>(n, x, y, z, vx, vy, vz, mass);
  hipDeviceSynchronize();
#else
  initialCondition_host(n, x, y, z, vx, vy, vz, mass);
#endif

  /*
   *  Use hipDeviceSetLimit() to change the buffer size of printf
   *   used in kernel functions to solve the problem encountered before:
   *    cannot print more than 4096 lines of data using printf
   *
   */
  hipDeviceSetLimit(hipLimitPrintfFifoSize, n * BUFFERSIZE);

  /*  Start looping steps from first step to mstep  */
  for(int i = 0; i < mstep; i++){
    if(i % nout == 0)
      printstate<<< grids, threads >>>(n, x, y, z, vx, vy, vz);
    hipDeviceSynchronize();

    /*
     *  Use hipDeviceSynchronize() to wait till all blocks of threads
     *   finish running the kernel functions
     *  Since between each accel() is called, the position of each particle
     *   is updated, which affect the second accel() calls, we need sychronize
     *   in the middle
     *
     */
    accel_3_body<<< grids, threads >>>(n, x, y, z, vx, vy, vz, mass, dt);
    hipDeviceSynchronize();
    leapstep<<< grids, threads >>>(n, x, y, z, vx, vy, vz, dt);
    hipDeviceSynchronize();
    accel_3_body<<< grids, threads >>>(n, x, y, z, vx, vy, vz, mass, dt);
    hipDeviceSynchronize();
  }
  if(mstep % nout == 0)
    printstate<<< grids, threads >>>(n, x, y, z, vx, vy, vz);
  hipDeviceSynchronize();

  // After finishing, free the allocated memory
  hipFree(x);
  hipFree(y);
  hipFree(z);
  hipFree(vx);
  hipFree(vy);
  hipFree(vz);
  hipFree(mass);

  // Exit the current thread
  hipDeviceReset();
  return 0;
}

/*
 *  Functions Implmenetation Section
 *
 */
__global__ void initialConditions(int n, double* x, double* y, double* z, double* vx, double* vy, double* vz, double* mass){
  /*  TODO    */
}

void rotate(double* x, double* y, double *z, double n1, double n2, double n3, double theta){

   double tmpx, tmpy, tmpz;
   double a, c, s, sigma;

   sigma = -theta;
   c = cos(sigma);
   s = sin(sigma);
   a = 1 - cos(sigma);


  tmpx = ( a * n1 * n1 + c ) * (*x) + ( a * n1 * n2 - s * n3) * (*y) + ( a * n1 * n3 + s * n2 ) * (*z);
  tmpy = ( a * n1 * n2 + s * n3) * (*x) + ( a * n2 * n2 + c) * (*y) + ( a * n2 * n3 - s * n1 ) * (*z);
  tmpz = ( a * n1 * n3 - s * n2) * (*x) + ( a * n2 * n3 + s * n1) * (*y) + ( a * n3 * n3 + c) * (*z);

  (*x) = tmpx;
  (*y) = tmpy;
  (*z) = tmpz;

}

void initialCondition_host(int n, double* x, double* y, double* z, double* vx, double* vy, double* vz, double* mass){
  srand(time(0));
  double *lx = (double*)malloc(n * sizeof(double));
  double *ly = (double*)malloc(n * sizeof(double));
  double *lz = (double*)malloc(n * sizeof(double));
  double *lvx = (double*)malloc(n * sizeof(double));
  double *lvy = (double*)malloc(n * sizeof(double));
  double *lvz = (double*)malloc(n * sizeof(double));
  double *lmass = (double*)malloc(n * sizeof(double));


  /*
   *  Setup mass of each particles (including center mass)
   *
   */
  int numofp1 = NUM_P_BASE * NUM_OF_RING_1 + (NUM_OF_RING_1 - 1) * INC_NUM_P * NUM_OF_RING_1 / 2 + 1;
  lmass[0] = MASS_1;    // Set the mass of center mass of 1st galaxy
  for(int i = 1; i < numofp1; i++){
    lmass[i] = PMASS;
  }
  lmass[numofp1] = MASS_2;
  for(int i = numofp1 + 1; i < n; i++){
    lmass[i] = PMASS;
  }

  /*
   *  Setup position of each particles
   *
   */
   // lx[0] = (double)rand() / RAND_MAX;
   // ly[0] = (double)rand() / RAND_MAX;
   // lz[0] = (double)rand() / RAND_MAX;
   // lvx[0] = lvy[0] = lvz[0] = 0.0;

   lx[0] = -RMAX / 2;
   ly[0] = 0.0;
   lz[0] = 0.0;
   lvx[0] = 0.0;
   lvy[0] = -sqrt(G * MASS_1 / (12 * RMIN));
   lvz[0] = 0.0;


   double cx = lx[0], cy = ly[0], cz = lz[0], cvx = lvx[0], cvy = lvy[0], cvz = lvz[0];
   double radius = RING_BASE_1;
   int count = 1;

   double omega = -PI / 6.0, sigma = PI / 3.0, norm;

   for(int i = 0; i < NUM_OF_RING_1; i++){
     int numOfP = NUM_P_BASE + INC_NUM_P * i;
     double piece = 2.0 * PI / numOfP;
     double velocity = sqrt(G * MASS_1 / radius);
     for(int j = 0; j < numOfP; j++){
       lx[count] = radius * cos(piece * j);
       ly[count] = radius * sin(piece * j);
       lz[count] = 0.0;
       lvx[count] = - velocity * sin(piece * j) * V_PARAMTER;
       lvy[count] = velocity * cos(piece * j) * V_PARAMTER;
       lvz[count] = 0.0;

#ifndef NR
       norm = sqrt(lx[count] * lx[count] + ly[count] * ly[count] + lz[count] * lz[count]);
       rotate(lx + count, ly + count, lz + count, cos(omega), sin(omega), 0, sigma);
#endif
       lx[count] += cx;
       ly[count] += cy;
       lz[count] += cz;

       /*
        *    TODO: set up initial condition for velocity
        */
#ifndef NR
       norm = sqrt(lvx[count] * lvx[count] + lvy[count] * lvy[count] + lvz[count] * lvz[count]);
       rotate(lvx + count, lvy + count, lvz + count, cos(omega), sin(omega), 0, sigma);
#endif
       lvx[count] += cvx;
       lvy[count] += cvy;
       lvz[count] += cvz;
       count++;
     }
     radius += INC_R_RING;
   }


   // lx[count] = lx[0] + radius * 3.0;
   // ly[count] = ly[0] + radius * 4.0;
   // lz[count] = lz[0];
   // lvx[count] = lvy[count] = lvz[count] = 0.0;

   lx[count] = RMAX / 2;
   ly[count] = 0.0;
   lz[count] = 0.0;
   lvx[count] = 0.0;
   lvy[count] = sqrt(G * MASS_2 / (12 * RMIN));
   lvz[count] = 0.0;

   cx = lx[count];
   cy = ly[count];
   cz = lz[count];
   cvx = lvx[count];
   cvy = lvy[count];
   cvz = lvz[count];
   count++;
   radius = RING_BASE_2;

   omega = - PI / 6.0;
   sigma = - PI / 3.0;
   for(int i = 0; i < NUM_OF_RING_2; i++){
    int numOfP = NUM_P_BASE + INC_NUM_P * i;
    double velocity = sqrt(G * MASS_2 / radius);
    double piece = 2.0 * PI / numOfP;
    for(int j = 0; j < numOfP; j++){
      lx[count] =  radius * cos(piece * j);
      ly[count] =  radius * sin(piece * j);
      lz[count] = 0.0;
      lvx[count] = - velocity * sin(piece * j) * V_PARAMTER;
      lvy[count] = velocity * cos(piece * j) * V_PARAMTER;
      lvz[count] = 0.0;
#ifndef NR
      norm = sqrt(lx[count] * lx[count] + ly[count] * ly[count] + lz[count] * lz[count]);
      rotate(lx + count, ly + count, lz + count, cos(omega), sin(omega), 0, sigma);
#endif
      lx[count] += cx;
      ly[count] += cy;
      lz[count] += cz;

      /*
       *  TODO: setup initial conditions for velocity
       */
#ifndef NR
      norm = sqrt(lvx[count] * lvx[count] + lvy[count] * lvy[count] + lvz[count] * lvz[count]);
      rotate(lvx + count, lvy + count, lvz + count, cos(omega), sin(omega), 0, sigma);
#endif
      lvx[count] += cvx;
      lvy[count] += cvy;
      lvz[count] += cvz;

      count++;
    }
    radius += INC_R_RING;
  }


  hipMemcpy(x, lx, (size_t) n * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(y, ly, (size_t) n * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(z, lz, (size_t) n * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(vx, lvx, (size_t) n * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(vy, lvy, (size_t) n * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(vz, lvz, (size_t) n * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(mass, lmass, (size_t) n * sizeof(double), hipMemcpyHostToDevice);
  free(lx);
  free(ly);
  free(lz);
  free(lvx);
  free(lvy);
  free(lvz);
  free(lmass);
}

__global__ void leapstep(int n, double *x, double *y, double *z, double *vx, double *vy, double *vz, double dt){
  const unsigned int serial = blockIdx.x * BLOCKSIZE + threadIdx.x;
  if(serial < n){
    x[serial] += dt * vx[serial];
    y[serial] += dt * vy[serial];
    z[serial] += dt * vz[serial];
  }
}


__global__ void accel(int n, double *x, double *y, double *z, double *vx, double *vy, double *vz, double* mass, double dt){
  const unsigned int serial = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int tdx = threadIdx.x;
  __shared__ double lx[BLOCKSIZE];
  __shared__ double ly[BLOCKSIZE];
  __shared__ double lz[BLOCKSIZE];

  if(serial < n){
    double ax = 0.0, ay = 0.0, az = 0.0, norm, thisX = x[serial], thisY = y[serial], thisZ = z[serial];
    for(int i = 0; i < gridDim.x; i++){
      // Copy data from main memory
      lx[tdx] = x[i * BLOCKSIZE + tdx];
      lz[tdx] = y[i * BLOCKSIZE + tdx];
      ly[tdx] = z[i * BLOCKSIZE + tdx];
      __syncthreads();

      // Accumulates the acceleration
      int itrSize = min(BLOCKSIZE, n - i * BLOCKSIZE);
      for(int j = 0; j < itrSize; j++){
        norm = pow(SOFTPARAMETER + pow(thisX - lx[j], 2) + pow(thisY - ly[j], 2) + pow(thisZ - lz[j], 2), 1.5);
        if(i * BLOCKSIZE + j != serial){
          ax += - G * mass[i * BLOCKSIZE + j] * (thisX - lx[j]) / norm;
          ay += - G * mass[i * BLOCKSIZE + j] * (thisY - ly[j]) / norm;
          az += - G * mass[i * BLOCKSIZE + j] * (thisZ - lz[j]) / norm;
        }
      }
    }

    // Updates velocities in each directions
    vx[serial] += 0.5 * dt * ax;
    vy[serial] += 0.5 * dt * ay;
    vz[serial] += 0.5 * dt * az;
  }
}

__global__ void accel_3_body(int n, double* x, double* y, double* z, double* vx, double* vy, double* vz, double* mass, double dt){
  /*
   *  Three body leapfrog: each particle is in a 3 body system with center mass of galaxy 1 and center mass of galaxy 2
   *    Because of SOFTPARAMETER, we dont need to determine if thread is computing against itself
   */
  const unsigned int serial = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numofp1 = NUM_P_BASE * NUM_OF_RING_1 + (NUM_OF_RING_1 - 1) * NUM_OF_RING_1 * INC_NUM_P / 2 + 1;
  double ax = 0.0, ay = 0.0, az = 0.0, norm1, norm2;
  double tempsp = (pow(pow(x[0] - x[numofp1], 2) + pow(y[0] - y[numofp1], 2) + pow(z[0] - z[numofp1], 2), 1.5) <= RMIN) ? 0.2 * RMIN : SOFTPARAMETER;
  double softparameter = (serial == 0 && serial == numofp1) ? tempsp : SOFTPARAMETER;
  norm1 = pow(softparameter + pow(x[serial] - x[0], 2) + pow(y[serial] - y[0], 2) + pow(z[serial] - z[0], 2), 1.5);
  norm2 = pow(softparameter + pow(x[serial] - x[numofp1], 2) + pow(y[serial] - y[numofp1], 2) + pow(z[serial] - z[numofp1], 2), 1.5);
  ax += -G * mass[0] * (x[serial] - x[0]) / norm1;
  ay += -G * mass[0] * (y[serial] - y[0]) / norm1;
  az += -G * mass[0] * (z[serial] - z[0]) / norm1;
  ax += -G * mass[numofp1] * (x[serial] - x[numofp1]) / norm2;
  ay += -G * mass[numofp1] * (y[serial] - y[numofp1]) / norm2;
  az += -G * mass[numofp1] * (z[serial] - z[numofp1]) / norm2;
  if(serial < n){
    vx[serial] += 0.5 * dt * ax;
    vy[serial] += 0.5 * dt * ay;
    vz[serial] += 0.5 * dt * az;
  }
}

__global__ void printstate(int n, double *x, double *y, double *z, double *vx, double *vy, double *vz){
  const unsigned int serial = blockIdx.x * blockDim.x + threadIdx.x;
  if(serial < n){
    printf("%d, %12.6f, %12.6f, %12.6f, %12.6f, %12.6f, %12.6f\n", serial, x[serial], y[serial], z[serial], vx[serial], vy[serial], vz[serial]);
  }
}

void printstate_host(int n, double *x, double *y, double *z, double *vx, double *vy, double *vz){
  double *lx = (double *)malloc(n * sizeof(double));
  double *ly = (double *)malloc(n * sizeof(double));
  double *lz = (double *)malloc(n * sizeof(double));
  double *lvx = (double *)malloc(n * sizeof(double));
  double *lvy = (double *)malloc(n * sizeof(double));
  double *lvz = (double *)malloc(n * sizeof(double));
  hipMemcpy(lx, x, (size_t)n * sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(ly, y, (size_t)n * sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(lz, z, (size_t)n * sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(lvx, vx, (size_t)n * sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(lvy, vy, (size_t)n * sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(lvz, vz, (size_t)n * sizeof(double), hipMemcpyDeviceToHost);
  for(int i = 0; i < n; i++){
    printf("%d, %12.6f, %12.6f, %12.6f, %12.6f, %12.6f, %12.6f\n", i, lx[i], ly[i], lz[i], lvx[i], lvy[i], lvz[i]);
  }
  free(lx);
  free(ly);
  free(lz);
  free(lvx);
  free(lvy);
  free(lvz);
}
